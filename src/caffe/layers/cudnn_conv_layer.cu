
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

// Binary added
#define SLOW_SECURITY_CHECKS

// Get L1 norm
template <typename Dtype>
inline __device__ Dtype getL1Norm(const Dtype* weightData, const int weightArea)
{
  // L1 norm
  auto l1Norm = Dtype(0);
  for (auto i = 0 ; i < weightArea ; i++)
    l1Norm += (weightData[i] < 0 ? -weightData[i] : weightData[i]);
  return l1Norm;
}

// \tilde{W} = alpha * B
//     alpha = ||W||_1 / n
//     n = c x h x w
//     B_i = sign(W_i)
template <typename Dtype>
__global__ void approximateWeightsGpu(Dtype* weightBinaryData, const Dtype* weightRealData, const int count,
                                      const int weightArea)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < count)
  {
    // Offset
    const auto offset = globalIdx * weightArea;
    const auto* weightRealDataOffset = &weightRealData[offset];
    auto* weightBinaryDataOffset = &weightBinaryData[offset];
    // XNOR-style
    // L1 norm & optimal alpha
    const auto alphaOptimal = getL1Norm(weightRealDataOffset, weightArea) / weightArea;
    // Update output
    for (auto i = 0 ; i < weightArea ; i++)
      weightBinaryDataOffset[i] = (weightRealDataOffset[i] < 0 ? -alphaOptimal : alphaOptimal);
  }
}

// Dtype data (integer, floating, etc.) into binary data
template <typename Dtype>
__global__ void dTypeToBinaryGpu(Dtype* binaryData, const Dtype* realData, const int count)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < count)
  {
    // realData[globalIdx] = max(-Dtype(1), min(Dtype(1), realData[globalIdx])); // When used as binarization
    binaryData[globalIdx] = (realData[globalIdx] < 0 ? Dtype(-1) : Dtype(1));
  }
}

// NxCxHxW --> Nx1xHxW
// output(n,1,h,w) = sum(abs(input(n,:,h,w)))
template <typename Dtype>
__global__ void addOverChannelsGpu(Dtype* outputData, const Dtype* inputData, const int bottomChannels,
                                   const int bottomWHArea)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < bottomWHArea)
  {
    auto& output = outputData[globalIdx];
    output = 0;
    for (auto i = 0 ; i < bottomChannels ; i++)
    {
      const auto value = inputData[globalIdx+i*bottomWHArea];
      output += (value < 0 ? -value : value);
    }
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::approximateInputGpu(Blob<Dtype>* bottom_binary_, Blob<Dtype>* matrix_A_,
  Blob<Dtype>* matrix_K_, const Blob<Dtype>* const matrix_one_over_chw,
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top, const int num, const int binaryOption) const
{
  // Full binary (weights + input)
  if (binaryOption == 3)
  {
    // Get binary input (bottom_binary_)
    const auto count = bottom_binary_->count();
    dTypeToBinaryGpu<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_binary_->mutable_gpu_data(), bottom[0]->gpu_data(), count);
    // Input (bottom) dependent
    const auto bottomChannels = bottom_binary_->shape(1);
    const auto bottomWHArea = bottom_binary_->count(2);
    const auto bottomNArea = bottomChannels * bottomWHArea;
    CHECK_EQ(bottomWHArea, matrix_A_->count(1));
    // Output (top) dependent
    const auto* const weightOneOverCHW = matrix_one_over_chw->gpu_data();
    const auto* const matrix_A_data = matrix_A_->gpu_data();
    // K matrix
    auto* matrix_K_data = matrix_K_->mutable_gpu_data();
    for (int n = 0; n < num; ++n)
    {
      // Get A matrix (matrix_A_)
      addOverChannelsGpu<<<CAFFE_GET_BLOCKS(bottomWHArea), CAFFE_CUDA_NUM_THREADS>>>(
        matrix_A_->mutable_gpu_data() + n*bottomWHArea, bottom[0]->gpu_data() + n*bottomNArea, bottomChannels,
        bottomWHArea);
      // Get K matrix (matrix_K_)
      CUDNN_CHECK(cudnnConvolutionForward(matrix_K_handle_,
        cudnn::dataType<Dtype>::one,
        matrix_A_desc_, matrix_A_data,
        matrix_one_filter_desc_, weightOneOverCHW,
        matrix_AK_conv_descs_,
        matrix_AK_fwd_algo_, workspace[0], matrix_AK_workspace_fwd_sizes_,
        cudnn::dataType<Dtype>::zero,
        matrix_K_desc_, matrix_K_data));
    }
  }
}

// Float data into binary data
template <typename Dtype>
__global__ void multiplyOverChannelsGpu(Dtype* outputData, const Dtype* multiplierData, const int topChannels,
                                        const int topWHArea)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < topWHArea)
    for (auto i = 0 ; i < topChannels ; i++)
      outputData[globalIdx+i*topWHArea] *= multiplierData[globalIdx];
}

// Binary weights = +-n - XNOR-style
template <typename Dtype>
__global__ void backwardNormalizeWeightsGpu(Dtype* bottomDiff, /*const Dtype* topDiff,*/ const Dtype* bottomData, const int count,
                                            const int weightArea)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < count)
  {
    // Offset
    const auto offset = globalIdx * weightArea;
    // const auto* topDiffOffset = &topDiff[offset];
    const auto* bottomDataOffset = &bottomData[offset];
    auto* bottomDiffOffset = &bottomDiff[offset];
    // XNOR-style
    // L1 norm
    // const auto l1Norm = getL1Norm(topDiffOffset, weightArea);
    // const auto l1Norm = getL1Norm(bottomDiffOffset, weightArea);
    const auto l1Norm = getL1Norm(bottomData, weightArea);
// bottomDiff or bottomData????????????????????????????????????????????
    // Update output
    const auto oneOverWeightArea = Dtype(1)/Dtype(weightArea);
    for (auto i = 0 ; i < weightArea ; i++)
      // bottomDiffOffset[i] = topDiffOffset[i] * oneOverWeightArea
      bottomDiffOffset[i] *= oneOverWeightArea
                          * (1 + l1Norm * max(-Dtype(1), min(Dtype(1), bottomDataOffset[i])));
  }
}

// XNOR-style
template <typename Dtype>
__global__ void backwardNormalizeInputGpu(Dtype* bottomDiff, /*const Dtype* topDiff,*/ const Dtype* bottomData, const int count,
                                          const Dtype oneOverWeightArea, const Dtype* matrixK, const int matrixKOffset)
{
  const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (globalIdx < count)
  {
    // bottomDiff[globalIdx] = topDiff[globalIdx] * (oneOverWeightArea
    bottomDiff[globalIdx] *= (oneOverWeightArea
                             + matrixK[globalIdx % matrixKOffset] * max(-Dtype(1), min(Dtype(1), bottomData[globalIdx])));
  }
}
// Binary added ended

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Binary added
  if (this->layer_param_.convolution_param().binary() > 0)
  {
    const auto binarizeWeightsThisFrame = (this->phase_ == TRAIN || !weight_initialized_);
    // TEST/TRAIN - First frame (initialization)
    if (!weight_initialized_)
    {
      CHECK_EQ(this->group_, 1) << "Binary conv net not implemented for !=1 groups.";
      CHECK_EQ(bottom.size(), 1) << "Binary conv net not implemented for !=1 bottoms.";
      weight_initialized_ = true;
      CHECK_GE(this->blobs_.size(), 1);
      CHECK_GT(this->blobs_[0]->shape().size(), 2u);
      weight_binary_.reset(new Blob<Dtype>());
      weight_binary_->Reshape(this->blobs_[0]->shape());
      if (this->layer_param_.convolution_param().binary() > 2)
      {
        // Blob initialization
        bottom_binary_.reset(new Blob<Dtype>());
        matrix_A_.reset(new Blob<Dtype>());
        matrix_K_.reset(new Blob<Dtype>());
        matrix_one_over_chw.reset(new Blob<Dtype>());
        // Blob reshape
        bottom_binary_->Reshape(bottom[0]->shape());
        matrix_A_->Reshape(bottom[0]->shape(0), 1, bottom[0]->shape(2), bottom[0]->shape(3));
        matrix_K_->Reshape(top[0]->shape(0), 1, top[0]->shape(2), top[0]->shape(3));
        matrix_one_over_chw->Reshape(bottom[0]->shape(0), 1, this->blobs_[0]->shape(2), this->blobs_[0]->shape(3));
        // Filling matrix_one_over_chw
        auto* inputOnes = matrix_one_over_chw->mutable_cpu_data();
        const auto bottomNArea = bottom_binary_->count(1);
        for (auto i = 0 ; i < matrix_one_over_chw->count() ; i++)
          inputOnes[i] = Dtype(1)/Dtype(bottomNArea);
      }
    }
    // 1 frame (if testing), every frame if train
    if (binarizeWeightsThisFrame)
    {
      const auto this_blobs_0 = this->blobs_[0];
      // // Option a - Weight = +-1
      // const auto count = this_blobs_0->count();
      // dTypeToBinaryGpu<<<CAFFE_GET_BLOCKS(count/binarizationArea), CAFFE_CUDA_NUM_THREADS>>>(
      //   weight_binary_->mutable_gpu_data(), this_blobs_0->mutable_gpu_data(), count);
      // Option b - Weight = +-n per w,h
      if (this->layer_param_.convolution_param().binary() == 1)
      {
        const auto count = this_blobs_0->count();
        const auto binarizationArea = this_blobs_0->count(2);
        const auto countReduced = count/binarizationArea;
        approximateWeightsGpu<<<CAFFE_GET_BLOCKS(countReduced), CAFFE_CUDA_NUM_THREADS>>>(
          weight_binary_->mutable_gpu_data(), this_blobs_0->gpu_data(), countReduced, binarizationArea);
      }
      // Option c - Weight = +-n per c,w,h
      else if (this->layer_param_.convolution_param().binary() > 1)
      {
        const auto binarizationArea = this_blobs_0->count(1);
        const auto countReduced = this_blobs_0->shape(0);
        approximateWeightsGpu<<<CAFFE_GET_BLOCKS(countReduced), CAFFE_CUDA_NUM_THREADS>>>(
          weight_binary_->mutable_gpu_data(), this_blobs_0->gpu_data(), countReduced, binarizationArea);
        // SECURITY CHECK
        #ifdef SLOW_SECURITY_CHECKS
          const auto cpuDataB = weight_binary_->cpu_data();
          const auto cpuDataW = this_blobs_0->cpu_data();
          for (auto i = 0 ; i < this_blobs_0->shape(0); i++)
          {
            auto counter = Dtype(0);
            for (auto j = 0 ; j < binarizationArea; j++)
              counter += std::abs(cpuDataW[j+i*binarizationArea]);
            counter /= binarizationArea;
            CHECK_EQ(counter, std::abs(cpuDataB[i*binarizationArea]));
            // std::cout << counter << " vs. " << cpuDataB[i*binarizationArea]
            //   << " vs. " << cpuDataB[i*binarizationArea+1] << " vs. " << cpuDataB[i*binarizationArea-1] << std::endl;
          }
        #endif

// for (auto asdf = 0 ; asdf < 20; asdf++)
//   std::cout << this_blobs_0->cpu_data()[asdf] << " ";
// std::cout << "\n";
// for (auto asdf = 0 ; asdf < 20; asdf++)
//   std::cout << weight_binary_->cpu_data()[asdf] << " ";
// std::cout << "\n\n" << std::endl;
      }
    }
    // Every frame
    approximateInputGpu(bottom_binary_.get(), matrix_A_.get(), matrix_K_.get(),
                        this->matrix_one_over_chw.get(), bottom, top, this->num_,
                        this->layer_param_.convolution_param().binary());
    // SECURITY CHECK
    #ifdef SLOW_SECURITY_CHECKS
      if (this->layer_param_.convolution_param().binary() == 3)
      {
        const auto bottomData = bottom[0]->cpu_data();
        const auto matrixAData = matrix_A_->cpu_data();
        // bottom_binary
        for (auto i = 0 ; i < bottom[0]->count(); i++)
          CHECK(bottomData[i] < 0
            ? bottom_binary_->cpu_data()[i] == -1
            : bottom_binary_->cpu_data()[i] == 1);
        // matrix_A_
        const auto whArea = bottom[0]->count(2);
        const auto cwhArea = bottom[0]->count(1);
        for (auto num = 0 ; num < bottom[0]->shape(0); num++)
        {
          for (auto xy = 0 ; xy < whArea; xy++)
          {
            auto counter = Dtype(0);
            for (auto c = 0 ; c < bottom[0]->shape(1); c++)
              counter += std::abs(bottomData[xy+c*whArea+num*cwhArea]);
            CHECK_EQ(counter, std::abs(matrixAData[xy+num*whArea]))
              << "Some values: " << bottomData[xy+num*cwhArea]
              << " " << bottomData[xy+1*whArea+num*cwhArea] << " " << bottomData[xy+2*whArea+num*cwhArea] << " " << bottomData[xy+3*whArea+num*cwhArea];
          }
        }
        // // matrix_K_
        // // No considered the borders to simplify operation
        // CHECK_EQ(this->blobs_[0]->count(2), 9) << "Slow security check only implemented for 3x3 convolutions.";
        // CHECK_EQ(matrix_K_->count(3), top[0]->count(3)) << "Slow security check only implemented for pad = 1 sceneario.";
        // CHECK_EQ(matrix_K_->shape(1), 1);
        // const auto yOffset = matrix_K_->count(3);
        // for (auto num = 0 ; num < top[0]->shape(0); num++)
        // {
        //   for (auto y = 1 ; y < top[0]->shape(2) - 1; y++)
        //   {
        //     for (auto x = 1 ; x < top[0]->shape(3) - 1; x++)
        //     {
        //       const auto baseIndex = num * matrix_A_->count(1) + y * yOffset + x;
        //       const auto counter = (matrixAData[-yOffset+baseIndex-1] + matrixAData[-yOffset+baseIndex] + matrixAData[-yOffset+baseIndex+1]
        //                             + matrixAData[baseIndex-1] + matrixAData[baseIndex] + matrixAData[baseIndex+1]
        //                             + matrixAData[yOffset+baseIndex-1] + matrixAData[yOffset+baseIndex] + matrixAData[yOffset+baseIndex+1]) / top[0]->count(1)
        //       / 2; // HACK TO MAKE IT WORK. WHY?????????!!!!!!!!!!!!
        //       const auto matrixKValue = matrix_K_->cpu_data()[num * matrix_K_->count(1) + y * matrix_K_->count(3) + x];
        //       if (y == 1 && x == 1)
        //       {
        //         if (num == 0)
        //           std::cout << "\n";
        //         std::cout << "n = " << num << "/" << top[0]->shape(0) << ": "
        //           << (std::abs(counter - matrixKValue)/matrixKValue, 1e-3) << ": " << counter << " vs. " << matrixKValue << std::endl;
        //       }
        //       // CHECK_EQ(counter/(2*top[0]->count(1)), matrix_K_->cpu_data()[num * top[0]->count(1) + y * top[0]->count(3) + x]);
        //       // CHECK_LE(std::abs(counter - matrixKValue)/matrixKValue, 1e-3) << counter << " vs. " << matrixKValue;
        //     }
        //   }
        // }
      }
    #endif
  }
  // Binary added end

  // const Dtype* weight = this->blobs_[0]->gpu_data(); // Binary commented
  // Binary added
  // const Dtype* weight = (this->layer_param_.convolution_param().binary() > 0 && this->phase_ == TRAIN
  //   ? weight_binary_->gpu_data() : this->blobs_[0]->gpu_data());
  const Dtype* weight = (this->layer_param_.convolution_param().binary() > 0
    ? weight_binary_->gpu_data() : this->blobs_[0]->gpu_data());
  // const Dtype* weight = (this->layer_param_.convolution_param().binary() > 0 && this->phase_ == TEST
  //   ? weight_binary_->gpu_data() : this->blobs_[0]->gpu_data());
  // Binary added ended
  for (int i = 0; i < bottom.size(); ++i) {
    // const Dtype* bottom_data = bottom[i]->gpu_data(); // Binary commented
    // Binary added
    const Dtype* bottom_data = (this->layer_param_.convolution_param().binary() > 2
      ? bottom_binary_->gpu_data() : bottom[i]->gpu_data());
    // Binary added ended
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

//       // Binary added
//       if (this->layer_param_.convolution_param().binary() > 2)
//       {
//         const auto topChannels = top[i]->shape(1);
//         const auto topWHArea = top[i]->count(2);
//         multiplyOverChannelsGpu<<<CAFFE_GET_BLOCKS(topWHArea), CAFFE_CUDA_NUM_THREADS>>>(
//           top_data + top_offset_ * g, matrix_K_->gpu_data(), topChannels, topWHArea);
// // for (auto asdf = 0 ; asdf < top[i]->count(); asdf++)
// // {
// // if (asdf % top[i]->count(1) == 0)
// // std::cout << "\n";
// // std::cout << top[i]->cpu_data()[asdf] << " ";
// // }
// // std::cout << std::endl;
// // for (auto asdf = 0 ; asdf < bottom_binary_->count(); asdf++)
// // for (auto asdf = 0 ; asdf < 20; asdf++)
// //   std::cout << bottom_binary_->cpu_data()[asdf] << " ";
// // std::cout << "\n";
// // for (auto asdf = 0 ; asdf < 20; asdf++)
// //   std::cout << bottom[i]->cpu_data()[asdf] << " ";
// // std::cout << "\n\n" << std::endl;
//       }
//       // Binary added end

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    // weight = this->blobs_[0]->gpu_data(); // Binary commented
    // Binary added
    // My binary way
    // weight = this->blobs_[0]->gpu_data();
    // Plain truncating
    weight = (this->layer_param_.convolution_param().binary() > 0
              ? weight_binary_->gpu_data() : this->blobs_[0]->gpu_data());
    // Binary added ended
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          // weight = this->blobs_[0]->gpu_data(); // Binary commented
          // Binary added
          weight = (this->layer_param_.convolution_param().binary() > 0
                    ? weight_binary_->gpu_data() : this->blobs_[0]->gpu_data());
          // Binary added ended
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
  // Binary added
  if (this->layer_param_.convolution_param().binary() > 0)
  {
    const auto binarizationArea = this->blobs_[0]->shape(2) * this->blobs_[0]->shape(3);
    // Binarized weights (XNOR-style)
    if (this->param_propagate_down_[0])
    {
      const auto count = this->blobs_[0]->count();
      // // Option a - Weight = +-1
      // // Do nothing
      // Option b - Weight = +-n
      const auto countReduced = count/binarizationArea;
      backwardNormalizeWeightsGpu<<<CAFFE_GET_BLOCKS(countReduced), CAFFE_CUDA_NUM_THREADS>>>(
        this->blobs_[0]->mutable_gpu_diff(), /*this->blobs_[0]->gpu_diff(),*/ this->blobs_[0]->gpu_data(),
        countReduced, binarizationArea);

      // // SECURITY CHECK
      // #ifdef SLOW_SECURITY_CHECKS
      //   const auto cpuDataW = this->blobs_[0]->cpu_data();
      //   const auto cpuDiffW = this->blobs_[0]->cpu_diff();
      //   const auto oneOverWeightArea = Dtype(1)/Dtype(binarizationArea);
      //   for (auto i = 0 ; i < this->blobs_[0]->count(); i++)
      //   {
      //     auto l1Norm = Dtype(0);
      //     for (auto j = 0 ; j < binarizationArea; j++)
      //       l1Norm += std::abs(cpuDataW[j+i*binarizationArea]);
      //     l1Norm /= binarizationArea;
      //     const auto diff = cpuDiffW[i] * oneOverWeightArea
      //                     * (1 + l1Norm * max(-Dtype(1), min(Dtype(1), cpuDataW[i])));
      //     CHECK_EQ(diff, std::abs(cpuDataW[i]));
      //     // std::cout << counter << " vs. " << cpuDataB[i*binarizationArea]
      //     //   << " vs. " << cpuDataB[i*binarizationArea+1] << " vs. " << cpuDataB[i*binarizationArea-1] << std::endl;
      //   }
      // #endif
    }
    // // Binarized activations (XNOR-style)
    // if (this->layer_param_.convolution_param().binary() == 3)
    // {
    //   for (int i = 0; i < top.size(); ++i)
    //   {
    //     // Gradient w.r.t. bottom data.
    //     if (propagate_down[i])
    //     {
    //       const auto count = bottom[i]->count();
    //       const Dtype oneOverWeightArea = Dtype(1) / Dtype(binarizationArea);
    //       const auto matrixKOffset = bottom[i]->count(2);
    //       backwardNormalizeInputGpu<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    //         bottom[i]->mutable_gpu_diff(), /*bottom[i]->gpu_diff(),*/ bottom[i]->gpu_data(),
    //         count, oneOverWeightArea, matrix_K_->gpu_data(), matrixKOffset);
    //     }
    //   }
    // }
  }
  // // Regularization - My binary way (guiding weights to 1)
  // if (this->layer_param_.convolution_param().binary() > 0) // Binary added
  // {
  //   if (this->param_propagate_down_[0]) {
  //     const auto lambda = 0.01f;
  //     const auto* const weight_real = this->blobs_[0]->cpu_data();
  //     auto* weight_real_diff = this->blobs_[0]->mutable_cpu_diff();
  //     for (auto index = 0 ; index < this->blobs_[0]->count() ; index++)
  //       weight_real_diff[index] += 2*lambda*(   weight_real[index] - (weight_real[index] < 0 ? -1 : 1)   );
  //   }
  // }
  // Binary added end
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
